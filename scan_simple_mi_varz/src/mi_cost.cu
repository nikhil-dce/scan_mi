#include "hip/hip_runtime.h"
#include "mi_cost.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <iostream>
#include <stdio.h>
#include <cfloat>
#include <boost/timer/timer.hpp>

#include <thrust/extrema.h>
#include <thrust/pair.h>
#include <thrust/device_ptr.h>

MI_ScanPoint *d_scanA, *d_scanB, *d_transformedScanB;
int *d_marginalHistA, *d_marginalHistB, *d_jointHistAB;

float *d_voxelDataA, *d_voxelDataB;
int *d_voxelPoints; float *d_voxelSumZ;

// CUDA compact 
bool *d_voxelPredicateArray;

mi_transform_t d_transform;
int size_a, size_b, voxel_size;

// resolution should be greater than 1
#define MAX_VOXELS_ALONG_DIM 500
#define MAX_VOXELS_ALONG_X 250
#define MAX_VOXELS_ALONG_Y 250
#define MAX_VOXELS_ALONG_Z 100
#define MAX_POINTS_IN_VOXEL 2000 // empirical value
#define MAX_VARIANCE 1
#define VARIANCE_BIN 5e-3
#define VAR_RANGE 200

int DEBUG = false;
const int resolution = 1;

struct xyz_minmax {
	float minX, minY, minZ, maxX, maxY, maxZ;
} minmaxA;

struct transform_to_entropy {
	int totalVoxels;

	transform_to_entropy(int _totalVoxels) : totalVoxels(_totalVoxels) {}

	__device__
	double operator()(const int voxels) {
		double p = (double) voxels / totalVoxels;
		
		if (p == 0)
			return 0;

		p = p * log(p);
		return p;

	}

};

struct expand_to_xyzminmax {
  
  __device__  
  xyz_minmax operator()(const MI_ScanPoint p){
    xyz_minmax result;
    result.minX = p.x;
    result.maxX = p.x;
    result.minY = p.y;
    result.maxY = p.y;
    result.minZ = p.z;
    result.maxZ = p.z;
    return result;
  }

};

struct minmax3_functor {
  
  __device__
  xyz_minmax operator()(const xyz_minmax a, const xyz_minmax b) {
    xyz_minmax result;
    result.minX = (a.minX < b.minX) ? a.minX:b.minX;
    result.maxX = (a.maxX > b.maxX) ? a.maxX:b.maxX;
    result.minY = (a.minY < b.minY) ? a.minY:b.minY;
    result.maxY = (a.maxY > b.maxY) ? a.maxY:b.maxY;
    result.minZ = (a.minZ < b.minZ) ? a.minZ:b.minZ;
    result.maxZ = (a.maxZ > b.maxZ) ? a.maxZ:b.maxZ;
    return result;
  }
};

struct transform_point {

	double r0, r1, r2, r3, r4, r5, r6, r7, r8;
	double t0, t1, t2;	

	transform_point(double _r0, double _r1, double _r2, 
					double _r3, double _r4, double _r5, 
					double _r6, double _r7, double _r8, 
					double _t0, double _t1, double _t2) :
					r0(_r0), r1(_r1), r2(_r2),
					r3(_r3), r4(_r4), r5(_r5),
					r6(_r6), r7(_r7), r8(_r8),
					t0(_t0), t1(_t1), t2(_t2) {}
 
 	__host__ __device__
  MI_ScanPoint operator()(MI_ScanPoint basePoint)
  {
	
	MI_ScanPoint transformed_p;	
	transformed_p.x = r0 * basePoint.x + r1 * basePoint.y + r2 * basePoint.z + t0;
	transformed_p.y = r3 * basePoint.x + r4 * basePoint.y + r5 * basePoint.z + t1;
	transformed_p.z = r6 * basePoint.x + r7 * basePoint.y + r8 * basePoint.z + t2;

	transformed_p.refc = basePoint.refc;

	return transformed_p;
  }
};

struct transform_predicate_to_integer {	
 	
 	__host__ __device__
  int operator()(bool predicate)
  {
	
	if (predicate)
		return 1;
	else
		return 0;
	
  }
};

__host__ __device__ int
get_voxel_index (float px, float py, float pz, int resolution) {

	int key = 0;

	float x = px;
	float y = py;
	float z = pz;

	int keyx = x / resolution;
	int keyy = y / resolution;
	int keyz = z / resolution;

	if (x < 0) 
		keyx -= 1;		

	if (y < 0) 
		keyy -= 1;
	
	if (z < 0) 
		keyz -= 1;		

	keyx += MAX_VOXELS_ALONG_X / 2;
	keyy += MAX_VOXELS_ALONG_Y / 2;
	keyz += MAX_VOXELS_ALONG_Z / 2;
	
	
	key = 	keyx * MAX_VOXELS_ALONG_Y * MAX_VOXELS_ALONG_Z +
			keyy * MAX_VOXELS_ALONG_Z + 
			keyz;			

	return key;
}

__host__ __device__ MI_ScanPoint
get_point_for_key (int key) {

	MI_ScanPoint p;

	int keyx = key / (MAX_VOXELS_ALONG_Y * MAX_VOXELS_ALONG_Z);

	int remain1 = key % (MAX_VOXELS_ALONG_Y * MAX_VOXELS_ALONG_Z);
	int keyy = remain1 / MAX_VOXELS_ALONG_Z;
	int remain2 = remain1 % MAX_VOXELS_ALONG_Z;

	int keyz = remain2;

	keyx -= MAX_VOXELS_ALONG_X / 2;
	keyy -= MAX_VOXELS_ALONG_Y / 2;
	keyz -= MAX_VOXELS_ALONG_Z / 2;

	p.x = keyx;
	p.y = keyy;
	p.z = keyz;
	return p;
}

__global__ void
voxelize_scan (MI_ScanPoint *d_scan, float *d_voxelSumZ, int *d_voxelPoints, int size, int resolution) {

	int tid = blockIdx.x * blockDim.x + threadIdx.x;

	if (tid < size) {

		MI_ScanPoint scanPoint = d_scan[tid];
		int voxelIndex = get_voxel_index(scanPoint.x, scanPoint.y, scanPoint.z, resolution);
		
		atomicAdd(d_voxelPoints + voxelIndex, 1);		

		float z = scanPoint.z;
		if (z < 0)
			z *= -1;

		atomicAdd(d_voxelSumZ + voxelIndex, z);
				
	}	
}

__global__ void 
compute_voxel_variance (MI_ScanPoint *d_scan, float *d_voxelSumZ, int *d_voxelPoints, float *d_voxelData, int size) {

	int tid = blockIdx.x * blockDim.x + threadIdx.x;

	if (tid < size) {

		MI_ScanPoint scanPoint = d_scan[tid];
		int voxelIndex = get_voxel_index(scanPoint.x, scanPoint.y, scanPoint.z, resolution);

		int totalPointsInVoxel = d_voxelPoints[voxelIndex];
		float mean = d_voxelSumZ[voxelIndex] / totalPointsInVoxel;			

		float z = scanPoint.z;
		if (z < 0)
			z *= -1;

		float meanDistance = z - mean;

		float xVar = (meanDistance * meanDistance) / totalPointsInVoxel;
		atomicAdd (d_voxelData + voxelIndex, xVar);		
	}

}

__global__ void
transform_voxelize_scan (MI_ScanPoint* d_scan, float* d_voxelSumZ, int *d_voxelPoints, MI_ScanPoint* d_transformed_scan, 
						int size, int resolution, double r0, 
						double r1, double r2, double r3, 
						double r4, double r5, double r6, 
						double r7, double r8, double t0, double t1, double t2) {

	int tid = blockIdx.x * blockDim.x + threadIdx.x;

	if (tid < size) {

		MI_ScanPoint basePoint = d_scan[tid];

		MI_ScanPoint scanPoint;	
		scanPoint.x = r0 * basePoint.x + r1 * basePoint.y + r2 * basePoint.z + t0;
		scanPoint.y = r3 * basePoint.x + r4 * basePoint.y + r5 * basePoint.z + t1;
		scanPoint.z = r6 * basePoint.x + r7 * basePoint.y + r8 * basePoint.z + t2;
		scanPoint.refc = basePoint.refc;

		d_transformed_scan[tid] = scanPoint;
		int voxelIndex = get_voxel_index(scanPoint.x, scanPoint.y, scanPoint.z, resolution);
		
		atomicAdd(d_voxelPoints + voxelIndex, 1);		

		float z = scanPoint.z;
		if (z < 0)
			z *= -1;

		atomicAdd(d_voxelSumZ + voxelIndex, z);
	}	
}

__global__ void
create_histogram (float* d_voxelDataA, float* d_voxelDataB, bool* predicate, 
				  int* d_marginalHistA, int* d_marginalHistB, int* d_jointHistAB, 
				  int size) {

	int tid = blockIdx.x * blockDim.x + threadIdx.x;

	if (tid < size) {
		
		if (predicate[tid]) { 
			
			// calculate histogram for only the overlappint region

			// tid - voxelIdentifier
			// 2 global reads
			float dataA = d_voxelDataA[tid];
			float dataB = d_voxelDataB[tid];

			int indexA = (dataA + VARIANCE_BIN / 2) / VARIANCE_BIN;
			int indexB = (dataB + VARIANCE_BIN / 2) / VARIANCE_BIN;

			int jointIndex = indexA * VAR_RANGE + indexB;
			// 2 global ATOMIC adds
			// Use shared memory here
			atomicAdd(d_marginalHistA+indexA, 1);
			atomicAdd(d_marginalHistB+indexB, 1);		
			atomicAdd(d_jointHistAB+jointIndex, 1);		

		}
	}	
}

__global__ void
setupPredicateForOverlappingRegion(bool *d_voxelPredicateArray, int size, int resolution,
								   float minX, float minY, float minZ, float maxX, float maxY, float maxZ) {

	int tid = blockIdx.x * blockDim.x + threadIdx.x;

	if (tid < size) {

		// check if tid in overlapping region
		MI_ScanPoint p = get_point_for_key(tid);

		if (p.x < minX || p.x >= maxX ||
			p.y < minY || p.y >= maxY ||
			p.z < minZ || p.z >= maxZ ) {

			// predicate false
			d_voxelPredicateArray[tid] = false;
			
		} else {
			// predicate true
			d_voxelPredicateArray[tid] = true;		
		}

	}

}

void
initializeDeviceData (std::vector<MI_ScanPoint> h_scanA, std::vector<MI_ScanPoint> h_scanB) {	
	
	size_a = h_scanA.size();
	size_b = h_scanB.size();
	
	voxel_size = MAX_VOXELS_ALONG_X*MAX_VOXELS_ALONG_Y*MAX_VOXELS_ALONG_Z;

	std::cout << "Initializing Data" << std::endl;
	std::cout << "SizeA: " << size_a << std::endl;
	std::cout << "SizeB: " << size_b << std::endl;

	boost::timer::cpu_timer timer;
	boost::timer::cpu_times elapsed;
	
	checkCudaErrors(hipMalloc(&d_scanA, sizeof(struct MI_ScanPoint) * size_a));
	checkCudaErrors(hipMalloc(&d_scanB, sizeof(struct MI_ScanPoint) * size_b));
	checkCudaErrors(hipMalloc(&d_transformedScanB, sizeof(MI_ScanPoint) * size_b)); 

	if (DEBUG) {
		elapsed = timer.elapsed();
		std::cout << "CPU Time: " << (elapsed.user + elapsed.system) / 1e9 << " seconds" << " Actual Time: " << elapsed.wall / 1e9 << " seconds" << std::endl;	
		std::cout << "Allocating voxel space " << std::endl;
	}

	// needed to temorarily store reflectivity values
	checkCudaErrors(hipMalloc(&d_voxelPoints, sizeof(int) * voxel_size));	// to store total points	
	checkCudaErrors(hipMalloc(&d_voxelSumZ, sizeof(float) * voxel_size));	//store sum
	checkCudaErrors(hipMalloc(&d_voxelDataA, sizeof(float) * voxel_size));	//store variances
	checkCudaErrors(hipMalloc(&d_voxelDataB, sizeof(float) * voxel_size));	

	checkCudaErrors(hipMalloc(&d_voxelPredicateArray, sizeof(bool) * voxel_size));		


	checkCudaErrors(hipMalloc(&d_marginalHistA, sizeof(int) * VAR_RANGE)); 
	checkCudaErrors(hipMalloc(&d_marginalHistB, sizeof(int) * VAR_RANGE)); 
	checkCudaErrors(hipMalloc(&d_jointHistAB, sizeof(int) * VAR_RANGE * VAR_RANGE));	

	elapsed = timer.elapsed();
	std::cout << "Device Allocation completed in time: " << (elapsed.user + elapsed.system) / 1e9 << " seconds" << " Actual Time: " << elapsed.wall / 1e9 << " seconds" << std::endl;		
	
	// loading scans in device	
	checkCudaErrors(hipMemcpy(d_scanA, &(h_scanA[0]), sizeof(MI_ScanPoint) * size_a, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_scanB, &(h_scanB[0]), sizeof(MI_ScanPoint) * size_b, hipMemcpyHostToDevice));
	checkCudaErrors(hipGetLastError());
	// checkCudaErrors(hipDeviceSynchronize());
	
	elapsed = timer.elapsed();
	std::cout << "Data initialized at time: " << (elapsed.user + elapsed.system) / 1e9 << " seconds" << " Actual Time: " << elapsed.wall / 1e9 << " seconds" << std::endl;	
}

void
preprocessScanA () {
	
	const int blockSize = 128;
	int gridSize = (size_a + blockSize) / blockSize;	

	// initialize voxel data to zero
	checkCudaErrors(hipMemset(d_voxelDataA, 0, sizeof(float) * voxel_size));

	checkCudaErrors(hipMemset(d_voxelSumZ, 0, sizeof(float) * voxel_size));
	
	// initialize temp voxel data to zero
	checkCudaErrors(hipMemset(d_voxelPoints, 0, sizeof(int) * voxel_size));	

	// voxelizing scan A
	// This stores the count of the number of A points in voxel in d_voxelDataA 
	voxelize_scan<<<gridSize, blockSize>>> (d_scanA, d_voxelSumZ, d_voxelPoints, size_a, resolution);
	compute_voxel_variance<<<gridSize, blockSize>>> (d_scanA, d_voxelSumZ, d_voxelPoints, d_voxelDataA, size_a);
	
	gridSize = (voxel_size + blockSize) / blockSize;
	
	// transform_reduce to compute minmax3 for scan A
	thrust::device_ptr<MI_ScanPoint> d_thrustScanA (d_scanA);	

	xyz_minmax limit_init;
	limit_init.minX = FLT_MAX;
	limit_init.maxX = FLT_MIN;
	limit_init.minY = FLT_MAX;
	limit_init.maxY = FLT_MIN;
	limit_init.minZ = FLT_MAX;
	limit_init.maxZ = FLT_MIN;	

	minmaxA = thrust::transform_reduce(d_thrustScanA, d_thrustScanA + size_a, expand_to_xyzminmax(), limit_init, minmax3_functor());
}

double
calculateMIForPose (mi_transform_t t) {
	
	if (DEBUG) 
		printTransform(t);

	const int blockSize = 128;
	int gridSize;

	boost::timer::cpu_timer timer;
	boost::timer::cpu_times elapsed;

	// Initialize scan B data
	checkCudaErrors(hipMemset(d_marginalHistA, 0, sizeof(int) * VAR_RANGE));
	checkCudaErrors(hipMemset(d_marginalHistB, 0, sizeof(int) * VAR_RANGE));
	checkCudaErrors(hipMemset(d_jointHistAB, 0, sizeof(int) * VAR_RANGE * VAR_RANGE));

	// initialize temp voxel data to zero
	checkCudaErrors(hipMemset(d_voxelDataB, 0, sizeof(float) * voxel_size));	
	checkCudaErrors(hipMemset(d_voxelPoints, 0, sizeof(int) * voxel_size));
	checkCudaErrors(hipMemset(d_voxelSumZ, 0, sizeof(float) * voxel_size));

	// no need for this
	// checkCudaErrors(hipMemset(d_voxelPredicateArray, false, sizeof(int) * MAX_POINTS_IN_VOXEL * MAX_POINTS_IN_VOXEL)); 
	if (DEBUG) {
		elapsed = timer.elapsed();	
		std::cout << "CPU Time: " << (elapsed.user + elapsed.system) / 1e9 << " seconds" << " Actual Time: " << elapsed.wall / 1e9 << " seconds" << std::endl;
		
		std::cout << "KernelLaunch - Get voxel hash for transformed scan B" << std::endl;
	}

	gridSize = (size_b + blockSize) / blockSize;
	transform_voxelize_scan<<<gridSize, blockSize>>> (d_scanB, d_voxelSumZ, d_voxelPoints, d_transformedScanB, size_b, resolution,	
																													t[0], t[1], t[2], 
																													t[4], t[5], t[6], 
																													t[8], t[9], t[10], 
																													t[3], t[7], t[11]);
	compute_voxel_variance<<<gridSize, blockSize>>> (d_transformedScanB, d_voxelSumZ, d_voxelPoints, d_voxelDataB, size_b);

	// gridSize = (voxel_size + blockSize) / blockSize;
	// //set_voxel_data<<<gridSize, blockSize>>> (d_tempVoxelData, d_voxelDataB, voxel_size);

	if (DEBUG) {
		elapsed = timer.elapsed();
		std::cout << "CPU Time: " << (elapsed.user + elapsed.system) / 1e9 << " seconds" << " Actual Time: " << elapsed.wall / 1e9 << " seconds" << std::endl;

		std::cout << "KernelLaunch - Transform reduce for minmax3 for transformed Scan B " << std::endl;
	}

	// thrust find minmax3
	// init xyz_minmax
	thrust::device_ptr<MI_ScanPoint> d_thrustTransformedScanB (d_transformedScanB);	
	thrust::device_ptr<int> d_thrustHistA (d_marginalHistA);
	thrust::device_ptr<int> d_thrustHistB (d_marginalHistB);
	thrust::device_ptr<int> d_thrustHistAB (d_jointHistAB);
	thrust::device_ptr<bool> d_thrustPredicateVoxel (d_voxelPredicateArray);

	xyz_minmax limit_init;
	limit_init.minX = FLT_MAX;
	limit_init.maxX = FLT_MIN;
	limit_init.minY = FLT_MAX;
	limit_init.maxY = FLT_MIN;
	limit_init.minZ = FLT_MAX;
	limit_init.maxZ = FLT_MIN;	
	
	xyz_minmax minmaxB = thrust::transform_reduce(d_thrustTransformedScanB, d_thrustTransformedScanB + size_b, expand_to_xyzminmax(), limit_init, minmax3_functor());

	if (DEBUG) {
		elapsed = timer.elapsed();
		std::cout << "CPU Time: " << (elapsed.user + elapsed.system) / 1e9 << " seconds" << " Actual Time: " << elapsed.wall / 1e9 << " seconds" << std::endl;	
		std::cout << "CPU calculation overlapping region from minmax3 A and B" << std::endl;
	}

	float minX, minY, minZ, maxX, maxY, maxZ;
	minX = std::max(minmaxA.minX, minmaxB.minX);
	maxX = std::min(minmaxA.maxX, minmaxB.maxX);
	minY = std::max(minmaxA.minY, minmaxB.minY);
	maxY = std::min(minmaxA.maxY, minmaxB.maxY);
	minZ = std::max(minmaxA.minZ, minmaxB.minZ);
	maxZ = std::min(minmaxA.maxZ, minmaxB.maxZ);

	if (DEBUG) {

		std::cout << "minmaxA" << std::endl;
		std::cout << "MinX" << minmaxA.minX << " MaxX: " << minmaxA.maxX << std::endl;
		std::cout << "MinY" << minmaxA.minY << " MaxY: " << minmaxA.maxY << std::endl;
		std::cout << "MinZ" << minmaxA.minZ << " MaxZ: " << minmaxA.maxZ << std::endl;		
		std::cout << "minmaxB" << std::endl;
		std::cout << "MinX" << minmaxB.minX << " MaxX: " << minmaxB.maxX << std::endl;
		std::cout << "MinY" << minmaxB.minY << " MaxY: " << minmaxB.maxY << std::endl;
		std::cout << "MinZ" << minmaxB.minZ << " MaxZ: " << minmaxB.maxZ << std::endl;		
		std::cout << "Overlap" << std::endl;
		std::cout << "MinX" << minX << " MaxX: " << maxX << std::endl;
		std::cout << "MinY" << minY << " MaxY: " << maxY << std::endl;
		std::cout << "MinZ" << minZ << " MaxZ: " << maxZ << std::endl;
	}

	minX = floor(minX);
	minY = floor(minY);
	minZ = floor(minZ);

	maxX = ceil(maxX);
	maxY = ceil(maxY);
	maxZ = ceil(maxZ);	

	if (DEBUG) {
		elapsed = timer.elapsed();
		std::cout << "CPU Time: " << (elapsed.user + elapsed.system) / 1e9 << " seconds" << " Actual Time: " << elapsed.wall / 1e9 << " seconds" << std::endl;

		std::cout << "Kernel Launch - SetupPredicate Array for voxels"	<< std::endl;	
	}

	// Not using compact algorithm for cuda for now
	// as the histogram calculation is not an expensive operation
	// Will launch the kernel for all voxels and only consdier the ones with true predicate for now
	gridSize = (voxel_size + blockSize) / blockSize;
	setupPredicateForOverlappingRegion<<<gridSize, blockSize>>> (d_voxelPredicateArray, voxel_size, 1, minX, minY, minZ, maxX, maxY, maxZ);
	
	if (DEBUG) {
		elapsed = timer.elapsed();
		std::cout << "CPU Time: " << (elapsed.user + elapsed.system) / 1e9 << " seconds" << " Actual Time: " << elapsed.wall / 1e9 << " seconds" << std::endl;
		std::cout << "Creating histograms from predicate and voxelMapping"	<< std::endl;	
	}	
	
	//std::cout << "Calulate overlapping region size using predicate transform_reduce" << std::endl;
	//int initSum = 0;
	//int numberOfVoxels = thrust::transform_reduce(d_thrustPredicateVoxel, d_thrustPredicateVoxel+voxel_size , transform_predicate_to_integer(), initSum, thrust::plus<int>());
	int numberOfVoxels = (maxX - minX) * (maxY - minY) * (maxZ - minZ);
	//elapsed = timer.elapsed();
	//std::cout << "CPU Time: " << (elapsed.user + elapsed.system) / 1e9 << " seconds" << " Actual Time: " << elapsed.wall / 1e9 << " seconds" << std::endl;	
	
	if (DEBUG) {

		// float *h_voxelA = (float*) malloc (sizeof(float) * voxel_size);		
		// float *h_voxelB = (float*) malloc (sizeof(float) * voxel_size);
		// bool *h_predicate = (bool*) malloc (sizeof(bool) * voxel_size);

		// checkCudaErrors(hipMemcpy(h_voxelA, d_voxelDataA, sizeof(float) * voxel_size, hipMemcpyDeviceToHost));		
		// checkCudaErrors(hipMemcpy(h_voxelB, d_voxelDataB, sizeof(float) * voxel_size, hipMemcpyDeviceToHost));		
		// checkCudaErrors(hipMemcpy(h_predicate, d_voxelPredicateArray, sizeof(bool) * voxel_size, hipMemcpyDeviceToHost));

		// saveFloatArray ("DEBUG/voxel_A.txt", h_voxelA, voxel_size);		
		// saveFloatArray ("DEBUG/voxel_B.txt", h_voxelB, voxel_size);	
		// savePredicate ("DEBUG/predicate.txt", h_predicate, voxel_size);
	}
	
	gridSize = (voxel_size + blockSize) / blockSize;
	create_histogram<<<gridSize, blockSize>>> (d_voxelDataA, d_voxelDataB, d_voxelPredicateArray, d_marginalHistA, d_marginalHistB, d_jointHistAB, voxel_size);

	if (DEBUG) {
		elapsed = timer.elapsed();
		std::cout << "CPU Time: " << (elapsed.user + elapsed.system) / 1e9 << " seconds" << " Actual Time: " << elapsed.wall / 1e9 << " seconds" << std::endl;	
		std::cout << "Kernel launch - Transform to entropy and reduce " << std::endl;
	}

	double init_marginal = 0.;
	// Convert this into single kernel
	// Entropy of A on the other hand will change as the overlapping region changes
	double marginalEntropyA = thrust::transform_reduce(d_thrustHistA, d_thrustHistA+VAR_RANGE , transform_to_entropy(numberOfVoxels), init_marginal, thrust::plus<double>());
	double marginalEntropyB = thrust::transform_reduce(d_thrustHistB, d_thrustHistB+VAR_RANGE , transform_to_entropy(numberOfVoxels), init_marginal, thrust::plus<double>());
	double jointEntropyAB   = thrust::transform_reduce(d_thrustHistAB, d_thrustHistAB+(VAR_RANGE*VAR_RANGE) , transform_to_entropy(numberOfVoxels), init_marginal, thrust::plus<double>());

	checkCudaErrors(hipGetLastError());	

	double mi = -(marginalEntropyA + marginalEntropyB - jointEntropyAB);

	if (DEBUG) {
		elapsed = timer.elapsed();
		std::cout << "CPU Time: " << (elapsed.user + elapsed.system) / 1e9 << " seconds" << " Actual Time: " << elapsed.wall / 1e9 << " seconds" << std::endl;	

		std::cout << "Total voxels: " << numberOfVoxels << std::endl;
		std::cout << "Entropy A: " << marginalEntropyA << std::endl;
		std::cout << "Entropy B: " << marginalEntropyB << std::endl;
		std::cout << "Joint Entropy AB: " << jointEntropyAB << std::endl;
		std::cout << "MI: " << mi << std::endl;
	}

	
	return mi;	
}

void
freeDeviceData() {
	hipFree (d_scanA);
	hipFree (d_transformedScanB);
	hipFree (d_scanB);
	hipFree (d_voxelDataA);
	hipFree (d_voxelDataB);
	hipFree (d_marginalHistA);
	hipFree (d_marginalHistB);
	hipFree (d_jointHistAB);
	hipFree (d_voxelPredicateArray);
	hipFree (d_voxelSumZ);
	hipFree (d_voxelPoints);
}

void
setDebug(bool _debug) {
	DEBUG = _debug;
}
